#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <hiprand/hiprand_kernel.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <thrust/device_vector.h>

const int COUNT_OF_THREADS = 128;
const int COUNT_OF_DOTS = 10000000;
const double a = 0;
const double b = M_PI / 2.;
const double yMin = 0;
const double yMax = 1;

double fI() { return sin(b) - sin(a); }

__global__ void mcKernel(const int countOfIterations, int* counts, const double a, const double b, const double yMin, const double yMax){
    int i = threadIdx.x;
    hiprandStatePhilox4_32_10_t  state;
    hiprand_init(i, 0, 0, &state);
    for (int j = 0; j < countOfIterations; j++)
    {
        double2 dot = hiprand_uniform2_double(&state);
        if (dot.y * yMax - yMin <= cos(dot.x * b - a))
            counts[i]++;
    }
}

int main()
{
    hipSetDevice(0);
    thrust::device_vector<int> counts(COUNT_OF_THREADS);
    int* rawCounts = thrust::raw_pointer_cast(counts.data());
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventRecord(start, 0);
    dim3 threadsPerBlock(COUNT_OF_THREADS);
    mcKernel <<<1, threadsPerBlock >> > (COUNT_OF_DOTS / COUNT_OF_THREADS, rawCounts, a, b, yMin, yMax);
    int countOfInnerPoints = thrust::reduce(counts.begin(), counts.end(), (int)0, thrust::plus<int>());
    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Calculated integral: %f;\nPrecise integral: %f.\n", (b - a) * (yMax - yMin) * countOfInnerPoints / COUNT_OF_DOTS, fI());
    printf("Time of calculation: %f seconds\n", elapsedTime / 1000);
    hipDeviceReset();
    system("pause");
    return 0;
}
